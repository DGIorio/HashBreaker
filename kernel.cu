#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <string>
#include <cstdint>
#include <cmath>
#include <stdexcept>
#include <chrono>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <map>
#include <algorithm>
#include <limits>

// --- Compile-Time Limit ---
// Max *TOTAL* string length the compiled code can handle in buffers.
constexpr int MAX_TOTAL_LEN_COMPILE_LIMIT = 80;
constexpr int MAX_VARIABLE_LEN_COMPILE_LIMIT = 14;


// --- Default Configuration Values ---
// These are used if the config file doesn't provide a value.
const std::string DEFAULT_CHARSET = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ_012";
constexpr int DEFAULT_MAX_VARIABLE_LEN = 8;
constexpr int DEFAULT_MAX_RESULTS_BUFFER = 4096;
constexpr int DEFAULT_THREADS_PER_BLOCK = 256;
constexpr int DEFAULT_PREFERRED_BLOCKS = 8192;
constexpr int DEFAULT_START_STRING_LEN = 1;
const std::string DEFAULT_OUTPUT_FILENAME = "found_crc_matches.txt";
const std::string DEFAULT_PREFIX = "";
const std::string DEFAULT_SUFFIX = "";
const std::string DEFAULT_CONTAINS = "";

// Struct to hold loaded configuration
struct ConfigData {
    std::string charset = DEFAULT_CHARSET;
    int max_results_buffer = DEFAULT_MAX_RESULTS_BUFFER;
    int threads_per_block = DEFAULT_THREADS_PER_BLOCK;
    int preferred_blocks = DEFAULT_PREFERRED_BLOCKS;
    std::string output_filename = DEFAULT_OUTPUT_FILENAME;
    std::string prefix = DEFAULT_PREFIX;
    std::string suffix = DEFAULT_SUFFIX;
    std::string contains = DEFAULT_CONTAINS;
    int start_string_len = DEFAULT_START_STRING_LEN;
    int max_variable_len = DEFAULT_MAX_VARIABLE_LEN;
    // Derived values
    unsigned long long strings_per_launch = 0;
    int prefix_len = 0;
    int suffix_len = 0;
    int contains_len = 0;
};


// Struct to hold results
struct FoundMatch {
    uint32_t target_hash;
    // Using the TOTAL compile-time limit for safety
    char found_string[MAX_TOTAL_LEN_COMPILE_LIMIT + 1];
};

// Helper macro for CUDA error checking
#define CHECK_CUDA_ERROR(call)                                          \
do {                                                                    \
    hipError_t err = call;                                             \
    if (err != hipSuccess) {                                           \
        fprintf(stderr, "CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        hipDeviceReset();                                              \
        throw std::runtime_error(hipGetErrorString(err));              \
    }                                                                   \
} while (0)


// --- Device Code ---

// Constant memory for CRC table
__constant__ uint32_t CRC32_TABLE_DEVICE[256] = {
    0x00000000, 0x04C11DB7, 0x09823B6E, 0x0D4326D9, 0x130476DC, 0x17C56B6B, 0x1A864DB2, 0x1E475005,
    0x2608EDB8, 0x22C9F00F, 0x2F8AD6D6, 0x2B4BCB61, 0x350C9B64, 0x31CD86D3, 0x3C8EA00A, 0x384FBDBD,
    0x4C11DB70, 0x48D0C6C7, 0x4593E01E, 0x4152FDA9, 0x5F15ADAC, 0x5BD4B01B, 0x569796C2, 0x52568B75,
    0x6A1936C8, 0x6ED82B7F, 0x639B0DA6, 0x675A1011, 0x791D4014, 0x7DDC5DA3, 0x709F7B7A, 0x745E66CD,
    0x9823B6E0, 0x9CE2AB57, 0x91A18D8E, 0x95609039, 0x8B27C03C, 0x8FE6DD8B, 0x82A5FB52, 0x8664E6E5,
    0xBE2B5B58, 0xBAEA46EF, 0xB7A96036, 0xB3687D81, 0xAD2F2D84, 0xA9EE3033, 0xA4AD16EA, 0xA06C0B5D,
    0xD4326D90, 0xD0F37027, 0xDDB056FE, 0xD9714B49, 0xC7361B4C, 0xC3F706FB, 0xCEB42022, 0xCA753D95,
    0xF23A8028, 0xF6FB9D9F, 0xFBB8BB46, 0xFF79A6F1, 0xE13EF6F4, 0xE5FFEB43, 0xE8BCCD9A, 0xEC7DD02D,
    0x34867077, 0x30476DC0, 0x3D044B19, 0x39C556AE, 0x278206AB, 0x23431B1C, 0x2E003DC5, 0x2AC12072,
    0x128E9DCF, 0x164F8078, 0x1B0CA6A1, 0x1FCDBB16, 0x018AEB13, 0x054BF6A4, 0x0808D07D, 0x0CC9CDCA,
    0x7897AB07, 0x7C56B6B0, 0x71159069, 0x75D48DDE, 0x6B93DDDB, 0x6F52C06C, 0x6211E6B5, 0x66D0FB02,
    0x5E9F46BF, 0x5A5E5B08, 0x571D7DD1, 0x53DC6066, 0x4D9B3063, 0x495A2DD4, 0x44190B0D, 0x40D816BA,
    0xACA5C697, 0xA864DB20, 0xA527FDF9, 0xA1E6E04E, 0xBFA1B04B, 0xBB60ADFC, 0xB6238B25, 0xB2E29692,
    0x8AAD2B2F, 0x8E6C3698, 0x832F1041, 0x87EE0DF6, 0x99A95DF3, 0x9D684044, 0x902B669D, 0x94EA7B2A,
    0xE0B41DE7, 0xE4750050, 0xE9362689, 0xEDF73B3E, 0xF3B06B3B, 0xF771768C, 0xFA325055, 0xFEF34DE2,
    0xC6BCF05F, 0xC27DEDE8, 0xCF3ECB31, 0xCBFFD686, 0xD5B88683, 0xD1799B34, 0xDC3ABDED, 0xD8FBA05A,
    0x690CE0EE, 0x6DCDFD59, 0x608EDB80, 0x644FC637, 0x7A089632, 0x7EC98B85, 0x738AAD5C, 0x774BB0EB,
    0x4F040D56, 0x4BC510E1, 0x46863638, 0x42472B8F, 0x5C007B8A, 0x58C1663D, 0x558240E4, 0x51435D53,
    0x251D3B9E, 0x21DC2629, 0x2C9F00F0, 0x285E1D47, 0x36194D42, 0x32D850F5, 0x3F9B762C, 0x3B5A6B9B,
    0x0315D626, 0x07D4CB91, 0x0A97ED48, 0x0E56F0FF, 0x1011A0FA, 0x14D0BD4D, 0x19939B94, 0x1D528623,
    0xF12F560E, 0xF5EE4BB9, 0xF8AD6D60, 0xFC6C70D7, 0xE22B20D2, 0xE6EA3D65, 0xEBA91BBC, 0xEF68060B,
    0xD727BBB6, 0xD3E6A601, 0xDEA580D8, 0xDA649D6F, 0xC423CD6A, 0xC0E2D0DD, 0xCDA1F604, 0xC960EBB3,
    0xBD3E8D7E, 0xB9FF90C9, 0xB4BCB610, 0xB07DABA7, 0xAE3AFBA2, 0xAAFBE615, 0xA7B8C0CC, 0xA379DD7B,
    0x9B3660C6, 0x9FF77D71, 0x92B45BA8, 0x9675461F, 0x8832161A, 0x8CF30BAD, 0x81B02D74, 0x857130C3,
    0x5D8A9099, 0x594B8D2E, 0x5408ABF7, 0x50C9B640, 0x4E8EE645, 0x4A4FFBF2, 0x470CDD2B, 0x43CDC09C,
    0x7B827D21, 0x7F436096, 0x7200464F, 0x76C15BF8, 0x68860BFD, 0x6C47164A, 0x61043093, 0x65C52D24,
    0x119B4BE9, 0x155A565E, 0x18197087, 0x1CD86D30, 0x029F3D35, 0x065E2082, 0x0B1D065B, 0x0FDC1BEC,
    0x3793A651, 0x3352BBE6, 0x3E119D3F, 0x3AD08088, 0x2497D08D, 0x2056CD3A, 0x2D15EBE3, 0x29D4F654,
    0xC5A92679, 0xC1683BCE, 0xCC2B1D17, 0xC8EA00A0, 0xD6AD50A5, 0xD26C4D12, 0xDF2F6BCB, 0xDBEE767C,
    0xE3A1CBC1, 0xE760D676, 0xEA23F0AF, 0xEEE2ED18, 0xF0A5BD1D, 0xF464A0AA, 0xF9278673, 0xFDE69BC4,
    0x89B8FD09, 0x8D79E0BE, 0x803AC667, 0x84FBDBD0, 0x9ABC8BD5, 0x9E7D9662, 0x933EB0BB, 0x97FFAD0C,
    0xAFB010B1, 0xAB710D06, 0xA6322BDF, 0xA2F33668, 0xBCB4666D, 0xB8757BDA, 0xB5365D03, 0xB1F740B4
};

// Device function for CRC32 calculation
__device__ uint32_t calculate_custom_crc32(const char* data, int length) {
    if (length <= 0) { return 0; }
    uint32_t result = 0;
    if (length < 4) {
        result = 0xFFFFFFFF;
        for (int i = 0; i < length; ++i) {
            uint8_t index = (result ^ static_cast<uint8_t>(data[i])) & 0xFF;
            result = (result >> 8) ^ CRC32_TABLE_DEVICE[index];
        }
        result = ~result; // Final XOR
    }
    else {
        uint32_t initial_value = (static_cast<uint32_t>(static_cast<uint8_t>(data[0])) << 24) |
            (static_cast<uint32_t>(static_cast<uint8_t>(data[1])) << 16) |
            (static_cast<uint32_t>(static_cast<uint8_t>(data[2])) << 8) |
            static_cast<uint32_t>(static_cast<uint8_t>(data[3]));
        result = ~initial_value; // Initialize with NOT of first 4 bytes
        for (int i = 4; i < length; ++i) {
            result = ((result << 8) | static_cast<uint32_t>(static_cast<uint8_t>(data[i]))) ^ CRC32_TABLE_DEVICE[(result >> 24) & 0xFF];
        }
        result = ~result; // Apply final NOT
    }
    return result & 0xFFFFFFFF; // Mask
}

// Device-side string search
__device__ const char* dev_strstr(const char* haystack, int haystack_len, const char* needle, int needle_len) {
    if (!needle || needle_len <= 0) return haystack;
    if (!haystack || haystack_len < needle_len) return nullptr;

    for (int i = 0; i <= haystack_len - needle_len; ++i) {
        int j = 0;
        volatile const char* h_ptr = haystack + i;
        volatile const char* n_ptr = needle;
        while (j < needle_len && h_ptr[j] == n_ptr[j]) {
            j++;
        }
        if (j == needle_len) {
            return haystack + i; // Found
        }
    }
    return nullptr; // Not found
}

// CUDA Kernel (uses compile-time limit for arrays)
__global__ void crack_multi_crc32_kernel(
    // Target Hashes
    const uint32_t* d_target_hashes, int num_targets,
    // Charset
    const char* d_charset, int charset_len,
    // Current Test Length & Index
    int string_len, unsigned long long start_index,
    // Constraints
    const char* d_prefix, int prefix_len,
    const char* d_suffix, int suffix_len,
    const char* d_contains, int contains_len,
    // Results
    FoundMatch* d_results, int* d_result_count, int max_results)
{
    // --- Calculate Variable Part Length ---
    int variable_len = string_len - prefix_len - suffix_len;
    // If constraints make this length impossible, exit thread
    if (variable_len < 0) {
        return;
    }

    // Calculate global thread ID relative to the variable part's search space
    unsigned long long tid = (unsigned long long)blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long current_variable_index = start_index + tid;


    // Candidate buffer sized by TOTAL compile-time limit
    char candidate[MAX_TOTAL_LEN_COMPILE_LIMIT + 1];

    // --- Construct Candidate String with Constraints ---
    // 1. Copy Prefix
    for (int i = 0; i < prefix_len; ++i) {
        candidate[i] = d_prefix[i];
    }

    // 2. Generate Variable Part based on index
    unsigned long long temp_index = current_variable_index;
    bool index_overflow = false; // Check if index is too large for this variable_len

    // Handle case where variable_len is 0 (string is just prefix + suffix)
    if (variable_len == 0) {
        if (current_variable_index > 0) { // Only index 0 is valid if variable_len is 0
            index_overflow = true;
        }
    }
    else {
        // Generate variable part chars from right to left
        for (int i = variable_len - 1; i >= 0; --i) {
            if (charset_len == 0) {
                index_overflow = true;
                break;
            }
            int char_idx = temp_index % charset_len;
            candidate[prefix_len + i] = d_charset[char_idx];
            temp_index /= charset_len;
        }
        // If temp_index is not 0, the original current_variable_index was too large
        if (temp_index != 0) {
            index_overflow = true;
        }
    }

    // If the index was out of bounds for the variable part, stop.
    if (index_overflow) {
        return;
    }

    // 3. Copy Suffix
    for (int i = 0; i < suffix_len; ++i) {
        candidate[prefix_len + variable_len + i] = d_suffix[i];
    }

    // 4. Null Terminate at the correct string_len
    candidate[string_len] = '\0';

    // --- Filter for CONTAINS constraint ---
    // Maybe it would be faster to ignore this check, and do it later in the results
    if (contains_len > 0) {
        if (dev_strstr(candidate, string_len, d_contains, contains_len) == nullptr) {
            return; // Does not contain the required substring, exit thread
        }
    }

    // --- CRC Calculation & Check (If all constraints passed) ---
    uint32_t calculated_crc = calculate_custom_crc32(candidate, string_len);

    // Check against every target hash (same as before)
    for (int j = 0; j < num_targets; ++j) {
        if (calculated_crc == d_target_hashes[j]) {
            int result_idx = atomicAdd(d_result_count, 1);
            if (result_idx < max_results) {
                d_results[result_idx].target_hash = d_target_hashes[j];
                // Copy the generated candidate string (up to string_len)
                for (int k = 0; k < string_len; ++k) {
                    d_results[result_idx].found_string[k] = candidate[k];
                }
                d_results[result_idx].found_string[string_len] = '\0';
            }
        }
    }
}


// --- Host Code ---

// Helper to trim whitespace from a string
std::string trim(const std::string& str) {
    size_t first = str.find_first_not_of(" \t\n\r\f\v");
    if (std::string::npos == first) {
        return str;
    }
    size_t last = str.find_last_not_of(" \t\n\r\f\v");
    return str.substr(first, (last - first + 1));
}

// Function to load configuration from file
bool loadConfig(const std::string& filename, ConfigData& config) {
    std::ifstream cFile(filename);
    if (!cFile.is_open()) {
        std::cerr << "Warning: Could not open config file: " << filename << std::endl;
        std::cerr << "Using default configuration values." << std::endl;
        config.prefix_len = config.prefix.length();
        config.suffix_len = config.suffix.length();
        config.contains_len = config.contains.length();
        config.strings_per_launch = (unsigned long long)config.threads_per_block * config.preferred_blocks;
        return true; // Prints a warning but allow proceeding with defaults
    }

    std::string line;
    int lineNum = 0;
    std::cout << "Loading configuration from: " << filename << std::endl;
    while (std::getline(cFile, line)) {
        lineNum++;
        line = trim(line);
        if (line.empty() || line[0] == '#') {
            continue; // Skip empty lines and comments
        }

        std::size_t eqPos = line.find('=');
        if (eqPos == std::string::npos) {
            std::cerr << "Warning: Invalid line format in config (missing '=') on line " << lineNum << ": \"" << line << "\". Skipping." << std::endl;
            continue;
        }

        std::string key = trim(line.substr(0, eqPos));
        std::string value = trim(line.substr(eqPos + 1));

        try {
            if (key == "CHARSET") {
                if (!value.empty()) config.charset = value;
                else std::cerr << "Warning: Empty value for CHARSET on line " << lineNum << ". Using default." << std::endl;
            }
            else if (key == "START_STRING_LEN") {
                int val = std::stoi(value);
                config.start_string_len = (val >= 1) ? val : DEFAULT_START_STRING_LEN;
                if (config.start_string_len != val) std::cerr << "Warning: Invalid START_STRING_LEN (" << val << ") on line " << lineNum << ". Using default/minimum (" << config.start_string_len << ")." << std::endl;
            }
            else if (key == "MAX_VARIABLE_LEN") {
                //int val = std::stoi(value);
                //config.max_variable_len = (val >= 0) ? val : DEFAULT_MAX_VARIABLE_LEN;
                //if (config.max_variable_len != val) std::cerr << "Warning: Invalid MAX_VARIABLE_LEN (" << val << ") on line " << lineNum << ". Using default/minimum (0)." << std::endl;

                int val = std::stoi(value);
                if (val > 0 && val <= MAX_VARIABLE_LEN_COMPILE_LIMIT) {
                    config.max_variable_len = val;
                }
                else if (val > MAX_VARIABLE_LEN_COMPILE_LIMIT) {
                    std::cerr << "Warning: MAX_VARIABLE_LEN (" << val << ") in config exceeds compile limit ("
                        << MAX_VARIABLE_LEN_COMPILE_LIMIT << "). Using compile limit." << std::endl;
                    config.max_variable_len = MAX_VARIABLE_LEN_COMPILE_LIMIT;
                }
                else {
                    std::cerr << "Warning: Invalid value for MAX_VARIABLE_LEN (" << val << ") on line " << lineNum << ". Using default (" << config.max_variable_len << ")." << std::endl;
                }
            }
            else if (key == "MAX_RESULTS_BUFFER") {
                int val = std::stoi(value);
                config.max_results_buffer = (val > 0) ? val : DEFAULT_MAX_RESULTS_BUFFER;
            }
            else if (key == "THREADS_PER_BLOCK") {
                int val = std::stoi(value);
                config.threads_per_block = (val > 0 && (val & (val - 1)) == 0 && val <= 1024) ? val : DEFAULT_THREADS_PER_BLOCK; // Basic power-of-2 check up to 1024
                if (config.threads_per_block != val) std::cerr << "Warning: Invalid THREADS_PER_BLOCK on line " << lineNum << ". Using default (" << config.threads_per_block << ")." << std::endl;
            }
            else if (key == "PREFERRED_BLOCKS") {
                int val = std::stoi(value);
                config.preferred_blocks = (val > 0) ? val : DEFAULT_PREFERRED_BLOCKS;
            }
            else if (key == "OUTPUT_FILENAME") {
                if (!value.empty()) config.output_filename = value;
                else std::cerr << "Warning: Empty value for OUTPUT_FILENAME on line " << lineNum << ". Using default." << std::endl;
            }
            else if (key == "PREFIX") {
                config.prefix = value;
            }
            else if (key == "SUFFIX") {
                config.suffix = value;
            }
            else if (key == "CONTAINS") {
                config.contains = value;
            }
            else {
                std::cerr << "Warning: Unknown configuration key on line " << lineNum << ": \"" << key << "\". Ignoring." << std::endl;
            }
        }
        catch (const std::invalid_argument& e) {
            std::cerr << "Warning: Invalid numeric value format on line " << lineNum << " for key '" << key << "': \"" << value << "\". Skipping." << std::endl;
        }
        catch (const std::out_of_range& e) {
            std::cerr << "Warning: Numeric value out of range on line " << lineNum << " for key '" << key << "': \"" << value << "\". Skipping." << std::endl;
        }
    }
    cFile.close();

    // Store lengths and calculate derived values
    config.prefix_len = config.prefix.length();
    config.suffix_len = config.suffix.length();
    config.contains_len = config.contains.length();
    config.strings_per_launch = (unsigned long long)config.threads_per_block * config.preferred_blocks;

    // --- Constraint Validation ---
    bool config_ok = true;
    int max_possible_total_len = config.prefix_len + config.suffix_len + config.max_variable_len;
    if (max_possible_total_len > MAX_TOTAL_LEN_COMPILE_LIMIT) {
        std::cerr << "Error: The maximum possible total string length calculated from\n"
            << "       Prefix (" << config.prefix_len << ") + Suffix (" << config.suffix_len
            << ") + Max Variable (" << config.max_variable_len << ") = " << max_possible_total_len << " characters\n"
            << "       exceeds the program's compiled buffer limit of "
            << MAX_TOTAL_LEN_COMPILE_LIMIT << " characters." << std::endl;
        std::cerr << "Solution: Shorten the Prefix, Suffix, or MAX_VARIABLE_LEN in the config file,\n"
            << "          OR recompile the program with a larger MAX_TOTAL_LEN_COMPILE_LIMIT value." << std::endl;
        std::cerr << "=======================================================================" << std::endl;
        config_ok = false;
    }

    // Validate start length against max possible length
    if (config.start_string_len > max_possible_total_len && config_ok) { // Only warn if other checks passed
        std::cerr << "\nWarning: START_STRING_LEN (" << config.start_string_len
            << ") is greater than the maximum possible total length (" << max_possible_total_len
            << ") allowed by constraints and MAX_VARIABLE_LEN. No strings will be tested." << std::endl;
    }
    // Validate start length against minimum possible length
    int min_possible_len = config.prefix_len + config.suffix_len;
    if (config.start_string_len < min_possible_len) {
        std::cerr << "\nWarning: START_STRING_LEN (" << config.start_string_len
            << ") is less than the minimum possible length (" << min_possible_len
            << ") required by Prefix/Suffix. Will start testing from length " << min_possible_len << "." << std::endl;
        // Adjusting the start later in main, this is just a warning
    }

    std::cout << "Configuration loaded." << (config_ok ? "" : " (With Errors!)") << std::endl;

    return config_ok;
}

// Function to read hashes from file
bool readHashesFromFile(const std::string& filename, std::vector<uint32_t>& targetHashes) {
    std::ifstream infile(filename);
    if (!infile.is_open()) {
        std::cerr << "Error: Could not open hash file: " << filename << std::endl;
        return false;
    }
    std::string line;
    int lineNum = 0;
    while (std::getline(infile, line)) {
        lineNum++;
        line = trim(line); // Use trim helper
        if (line.empty() || line[0] == '#') { continue; }
        try {
            size_t processed = 0;
            unsigned long long val = std::stoull(line, &processed, 16);
            if (processed != line.length() || val > UINT32_MAX) {
                std::cerr << "Warning: Invalid hash format or out of range on line " << lineNum << ": \"" << line << "\". Skipping." << std::endl;
                continue;
            }
            targetHashes.push_back(static_cast<uint32_t>(val));
        }
        catch (const std::invalid_argument& e) {
            std::cerr << "Warning: Invalid hash format on line " << lineNum << ": \"" << line << "\". Skipping." << std::endl;
        }
        catch (const std::out_of_range& e) {
            std::cerr << "Warning: Hash out of range on line " << lineNum << ": \"" << line << "\". Skipping." << std::endl;
        }
    }
    infile.close();
    return !targetHashes.empty();
}

// Function to write results to file
bool writeResultsToFile(const std::string& filename, const std::map<uint32_t,
    std::vector<std::string>>&results, const ConfigData& config, std::chrono::milliseconds duration,
    const char* gpuName) {
    std::ofstream outfile(filename);
    if (!outfile.is_open()) {
        std::cerr << "Error: Could not open output file for writing: " << filename << std::endl;
        return false;
    }

    outfile << "# CUDA CRC32 Cracker Results" << std::endl;
    outfile << "#----------------------------------------" << std::endl;
    outfile << "# Configuration Used:" << std::endl;
    outfile << "#   Charset: \"" << config.charset << "\"" << std::endl;
    outfile << "#   Start Total Length Tested: " << config.start_string_len << std::endl;
    outfile << "#   Max Variable Length Tested: " << config.max_variable_len << std::endl;
    outfile << "#   Compiled Total Length Buffer Limit: " << MAX_TOTAL_LEN_COMPILE_LIMIT << std::endl;
    outfile << "#   Prefix: \"" << config.prefix << "\"" << std::endl;
    outfile << "#   Suffix: \"" << config.suffix << "\"" << std::endl;
    outfile << "#   Contains: \"" << config.contains << "\"" << std::endl;
    outfile << "#   GPU Used: " << gpuName << std::endl;
    outfile << "#   Threads/Block: " << config.threads_per_block << std::endl;
    outfile << "#   Preferred Blocks/Launch: " << config.preferred_blocks << std::endl;
    outfile << "#----------------------------------------" << std::endl;
    outfile << "# Execution Time: " << std::fixed << std::setprecision(3) << duration.count() / 1000.0 << " seconds" << std::endl;
    outfile << "#----------------------------------------" << std::endl << std::endl;

    bool anyResults = false;
    for (const auto& pair : results) {
        uint32_t hash = pair.first;
        const auto& strings = pair.second;
        if (!strings.empty()) {
            anyResults = true;
            outfile << "Hash: 0x" << std::hex << std::setw(8) << std::setfill('0') << hash << std::dec << std::endl;
            for (const std::string& s : strings) {
                outfile << "  -> \"" << s << "\"" << std::endl;
            }
            outfile << std::endl;
        }
    }

    if (!anyResults) {
        outfile << "# No matches found for the provided hashes within the tested limits." << std::endl;
    }

    outfile.close();
    std::cout << "Results successfully written to: " << filename << std::endl;
    return true;
}


int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <config.ini> <hash_file.txt>" << std::endl;
        return 1;
    }
    std::string config_filename = argv[1];
    std::string hash_filename = argv[2];

    ConfigData config;
    std::vector<uint32_t> h_target_hashes;
    std::map<uint32_t, std::vector<std::string>> all_found_matches;

    // --- Load Config ---
    if (!loadConfig(config_filename, config)) {
        std::cerr << "Configuration loading failed or constraints invalid. Exiting." << std::endl;
        return 1; // Exit if config validation failed
    }

    // --- Read Hashes ---
    std::cout << "\nReading target hashes from: " << hash_filename << std::endl;
    if (!readHashesFromFile(hash_filename, h_target_hashes)) {
        std::cerr << "Error reading hashes or file is empty/invalid." << std::endl;
        return 1;
    }
    int num_targets = static_cast<int>(h_target_hashes.size());
    std::cout << "Successfully read " << num_targets << " target hashes." << std::endl;
    for (uint32_t hash : h_target_hashes) {
        all_found_matches[hash] = std::vector<std::string>(); // Initialize map entries
    }


    try {
        std::cout << "\n--- Effective Configuration ---" << std::endl;
        std::cout << "Charset: \"" << config.charset << "\" (Length: " << config.charset.length() << ")" << std::endl;
        std::cout << "Start Total String Length: " << config.start_string_len << std::endl;
        std::cout << "Max Variable Length: " << config.max_variable_len << std::endl;
        std::cout << "Compiled Total Length Buffer Limit: " << MAX_TOTAL_LEN_COMPILE_LIMIT << std::endl;
        std::cout << "Prefix Constraint: \"" << config.prefix << "\"" << std::endl;
        std::cout << "Suffix Constraint: \"" << config.suffix << "\"" << std::endl;
        std::cout << "Contains Constraint: \"" << config.contains << "\"" << std::endl;
        std::cout << "GPU Result Buffer Size: " << config.max_results_buffer << std::endl;
        std::cout << "Threads/Block: " << config.threads_per_block << std::endl;
        std::cout << "Preferred Blocks/Launch: " << config.preferred_blocks << std::endl;
        std::cout << "Target strings/launch: " << config.strings_per_launch << std::endl;
        std::cout << "Output File: " << config.output_filename << std::endl;
        std::cout << "-----------------------------" << std::endl;


        // --- GPU setup ---
        int deviceId;
        hipDeviceProp_t deviceProp;
        CHECK_CUDA_ERROR(hipGetDevice(&deviceId));
        CHECK_CUDA_ERROR(hipGetDeviceProperties(&deviceProp, deviceId));
        std::cout << "Using GPU: " << deviceProp.name << std::endl;
        if (deviceProp.major < 3) {
            std::cerr << "Warning: GPU Compute Capability might be too low (" << deviceProp.major << "." << deviceProp.minor << "). Requires >= 3.0 for atomicAdd." << std::endl;
        }


        // --- Device Memory Allocation (using config values) ---
        char* d_charset = nullptr;
        uint32_t* d_target_hashes = nullptr;
        FoundMatch* d_results = nullptr;
        int* d_result_count = nullptr;
        char* d_prefix = nullptr;
        char* d_suffix = nullptr;
        char* d_contains = nullptr;

        int charset_len = config.charset.length();

        CHECK_CUDA_ERROR(hipMalloc(&d_charset, charset_len * sizeof(char)));
        CHECK_CUDA_ERROR(hipMalloc(&d_target_hashes, num_targets * sizeof(uint32_t)));
        CHECK_CUDA_ERROR(hipMalloc(&d_results, config.max_results_buffer * sizeof(FoundMatch)));
        CHECK_CUDA_ERROR(hipMalloc(&d_result_count, sizeof(int)));
        // Allocate for constraints (even if empty, allocate minimally to avoid null pointers)
        CHECK_CUDA_ERROR(hipMalloc(&d_prefix, (config.prefix_len + 1) * sizeof(char)));
        CHECK_CUDA_ERROR(hipMalloc(&d_suffix, (config.suffix_len + 1) * sizeof(char)));
        CHECK_CUDA_ERROR(hipMalloc(&d_contains, (config.contains_len + 1) * sizeof(char)));


        // --- Host Memory for Results Batch ---
        std::vector<FoundMatch> h_results_batch(config.max_results_buffer);
        int h_total_processed_results = 0;


        // --- Data Transfers ---
        CHECK_CUDA_ERROR(hipMemcpy(d_charset, config.charset.c_str(), charset_len * sizeof(char), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_target_hashes, h_target_hashes.data(), num_targets * sizeof(uint32_t), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemset(d_result_count, 0, sizeof(int)));
        // Copy constraints (including null terminator for safety, though kernel uses length)
        CHECK_CUDA_ERROR(hipMemcpy(d_prefix, config.prefix.c_str(), (config.prefix_len + 1) * sizeof(char), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_suffix, config.suffix.c_str(), (config.suffix_len + 1) * sizeof(char), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(d_contains, config.contains.c_str(), (config.contains_len + 1) * sizeof(char), hipMemcpyHostToDevice));


        // --- Main Cracking Loop ---
        auto start_time = std::chrono::high_resolution_clock::now();
        bool buffer_overflow_warning = false;
        int min_possible_total_len = config.prefix_len + config.suffix_len; // Min length due to constraints

        std::cout << "\nStarting tests for variable lengths 0 to " << config.max_variable_len << "." << std::endl;
        if (config.start_string_len > min_possible_total_len) {
            std::cout << "(Will skip iterations until TOTAL length reaches " << config.start_string_len << ")" << std::endl;
        }

        // Loop through possible variable lengths
        for (int current_variable_len = 0; current_variable_len <= config.max_variable_len; ++current_variable_len) {
            // Calculate the total string length for this iteration
            int current_total_len = config.prefix_len + config.suffix_len + current_variable_len;

            if (current_total_len < config.start_string_len) {
                continue; // Skip this variable length, total length too short
            }

            // Check if total length exceeds compile limit (should be caught by loadConfig)
            if (current_total_len > MAX_TOTAL_LEN_COMPILE_LIMIT) {
                std::cerr << "\nInternal Error: Calculated total length " << current_total_len
                    << " exceeds compile limit " << MAX_TOTAL_LEN_COMPILE_LIMIT << ". Stopping." << std::endl;
                break; // Stop processing
            }

            std::cout << "\nTesting variable length: " << current_variable_len << " (Total length: " << current_total_len << ")" << std::endl;


            // Calculate total combinations for the VARIABLE part
            unsigned long long total_variable_combinations = 1;
            bool overflow_detected = false;
            if (current_variable_len > 0) { // Only calculate if there's a variable part
                unsigned long long check_val = 1;
                for (int i = 0; i < current_variable_len; ++i) {
                    if (charset_len > 0 && ULLONG_MAX / charset_len < check_val) {
                        total_variable_combinations = ULLONG_MAX;
                        overflow_detected = true;
                        break;
                    }
                    // Handle charset_len == 0
                    if (charset_len == 0) {
                        total_variable_combinations = 0; // No combinations possible
                        break;
                    }
                    check_val *= charset_len;
                }
                if (!overflow_detected && total_variable_combinations != 0) {
                    total_variable_combinations = check_val;
                }
            }
            else {
                total_variable_combinations = 1; // Only 1 combination if variable_len is 0
            }


            if (overflow_detected) {
                std::cout << "  Search space for length " << current_variable_len << " is extremely large." << std::endl;
            }
            else {
                std::cout << "  Variable combinations: " << total_variable_combinations << std::endl;
            }
            if (total_variable_combinations == 0 && current_variable_len > 0) {
                std::cout << "  Skipping variable length " << current_variable_len << " due to zero combinations (empty charset?)." << std::endl;
                continue;
            }


            // Batch loop
            unsigned long long current_start_index = 0;
            unsigned long long variable_strings_processed = 0;
            int h_result_count_device = 0;

            // Loop while index is valid for variable combinations
            while (current_start_index < total_variable_combinations || overflow_detected) {

                // Reset device counter if it's full
                if (h_result_count_device >= config.max_results_buffer) {
                    if (!buffer_overflow_warning) {
                        std::cerr << "\nWarning: GPU results buffer filled. Subsequent matches in this/prev batch might be missed until buffer cleared." << std::endl;
                        buffer_overflow_warning = true;
                    }
                    CHECK_CUDA_ERROR(hipMemset(d_result_count, 0, sizeof(int)));
                    h_total_processed_results += h_result_count_device;
                    h_result_count_device = 0;
                }

                // Determine grid size based on variable combinations
                unsigned long long strings_in_this_launch = config.strings_per_launch;
                if (!overflow_detected && current_start_index + strings_in_this_launch > total_variable_combinations) {
                    strings_in_this_launch = total_variable_combinations - current_start_index;
                }
                if (strings_in_this_launch == 0) break; // Should only happen if total_variable_combinations is 0 initially


                dim3 threadsPerBlock(config.threads_per_block);
                dim3 numBlocks((unsigned int)ceil((double)strings_in_this_launch / config.threads_per_block));

                // --- Kernel Launch (pass constraint pointers/lengths) ---
                crack_multi_crc32_kernel << <numBlocks, threadsPerBlock >> > (
                    d_target_hashes, num_targets,
                    d_charset, charset_len,
                    current_total_len,
                    current_start_index,
                    d_prefix, config.prefix_len,
                    d_suffix, config.suffix_len,
                    d_contains, config.contains_len,
                    d_results, d_result_count, config.max_results_buffer
                    );
                CHECK_CUDA_ERROR(hipGetLastError());

                variable_strings_processed += strings_in_this_launch;

                // Check for Results (Periodically)
                bool check_now = (variable_strings_processed % (config.strings_per_launch * 10) == 0) ||
                    (!overflow_detected && current_start_index + strings_in_this_launch >= total_variable_combinations) ||
                    (overflow_detected && variable_strings_processed % (config.strings_per_launch * 10) == 0);

                if (check_now) {
                    CHECK_CUDA_ERROR(hipDeviceSynchronize());
                    CHECK_CUDA_ERROR(hipMemcpy(&h_result_count_device, d_result_count, sizeof(int), hipMemcpyDeviceToHost));

                    int new_results_count = h_result_count_device;
                    if (new_results_count > 0) {
                        buffer_overflow_warning = false;
                        CHECK_CUDA_ERROR(hipMemcpy(h_results_batch.data(), d_results, new_results_count * sizeof(FoundMatch), hipMemcpyDeviceToHost));
                        for (int i = 0; i < new_results_count; ++i) {
                            all_found_matches[h_results_batch[i].target_hash].push_back(h_results_batch[i].found_string);
                        }
                        CHECK_CUDA_ERROR(hipMemset(d_result_count, 0, sizeof(int)));
                        h_total_processed_results += new_results_count;
                        h_result_count_device = 0;
                        //printf("\n[+] Found %d new matches. Total found so far: %d\n", new_results_count, h_total_processed_results);
                    }
                }

                // Progress Indicator
                if (!overflow_detected) {
                    double percent = (total_variable_combinations > 0) ? (double)variable_strings_processed / total_variable_combinations * 100.0 : 100.0;
                    printf("  Progress: %.2f%% (%llu / %llu) | Found: %d \r", percent, variable_strings_processed, total_variable_combinations, h_total_processed_results);
                }
                else {
                    printf("  Processed: %llu strings... | Found: %d \r", variable_strings_processed, h_total_processed_results);
                }
                fflush(stdout);

                // Update start index
                if (!overflow_detected) {
                    current_start_index += strings_in_this_launch;
                }
                else {
                    current_start_index += strings_in_this_launch;
                }
            } // End batch loop

            printf("\n");

            // Final check for results at the end of each length
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
            CHECK_CUDA_ERROR(hipMemcpy(&h_result_count_device, d_result_count, sizeof(int), hipMemcpyDeviceToHost));
            if (h_result_count_device > 0) {
                buffer_overflow_warning = false;
                CHECK_CUDA_ERROR(hipMemcpy(h_results_batch.data(), d_results, h_result_count_device * sizeof(FoundMatch), hipMemcpyDeviceToHost));
                for (int i = 0; i < h_result_count_device; ++i) {
                    all_found_matches[h_results_batch[i].target_hash].push_back(h_results_batch[i].found_string);
                }
                CHECK_CUDA_ERROR(hipMemset(d_result_count, 0, sizeof(int)));
                h_total_processed_results += h_result_count_device;
                //printf("[+] Found %d final matches for length %d. Total found so far: %d\n", h_result_count_device, current_len, h_total_processed_results);
                h_result_count_device = 0;
            }

        } // End length loop


        // --- Final Summary & Write Results ---
        auto end_time = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

        std::cout << "\n----------------------------------------" << std::endl;
        std::cout << "Cracking Complete!" << std::endl;
        std::cout << "Total execution time: " << duration.count() / 1000.0 << " seconds" << std::endl;
        std::cout << "Total matches found across all targets: " << h_total_processed_results << std::endl;
        std::cout << "----------------------------------------" << std::endl;

        writeResultsToFile(config.output_filename, all_found_matches, config, duration, deviceProp.name);


        // --- Cleanup ---
        std::cout << "\nCleaning up CUDA resources..." << std::endl;
        if (d_charset) hipFree(d_charset);
        if (d_target_hashes) hipFree(d_target_hashes);
        if (d_results) hipFree(d_results);
        if (d_result_count) hipFree(d_result_count);
        if (d_prefix) hipFree(d_prefix);
        if (d_suffix) hipFree(d_suffix);
        if (d_contains) hipFree(d_contains);
        CHECK_CUDA_ERROR(hipDeviceReset());

    }
    catch (const std::exception& e) {
        std::cerr << "\nRuntime Error: " << e.what() << std::endl;
        hipDeviceReset();
        return 1;
    }
    catch (...) {
        std::cerr << "\nAn unknown error occurred." << std::endl;
        hipDeviceReset();
        return 1;
    }

    return 0;
}